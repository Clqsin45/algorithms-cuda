#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/reduce.h>

#include <iostream>

#include "reduction1.h"
#include "reduction1_template.h"
#include "reduction2.h"
#include "reduction2_template.h"
#include "reduction3.h"
#include "reduction3_template.h"

typedef void (*pfnReduction)(int*, int*, const int*, size_t, int, int);

#define NUM_COUNT (1024 * 1024 * 1)
#define NUM_THREADS (1024)
#define MAX_BLOCKS (1024)

double run(int cIterations, int *answer, int *partial, const int *in, const size_t N, 
    const int numBlocks, int numThreads, pfnReduction func)
{
    hipEvent_t start, stop;
    checkCudaErrors( hipEventCreate(&start) );
    checkCudaErrors( hipEventCreate(&stop) );
    hipEventRecord(start);

    for (int i = 0; i < cIterations; ++i)
    {
        func(answer, partial, in, N, numBlocks, numThreads);
    }

    hipEventRecord(stop);
    checkCudaErrors( hipDeviceSynchronize() );
    float time = 0;
    hipEventElapsedTime(&time, start, stop);

    return time;
}

int main(int argc, char const *argv[])
{
    int blocks = (NUM_COUNT + NUM_THREADS - 1) / NUM_THREADS;
    if (blocks > MAX_BLOCKS) blocks = MAX_BLOCKS;

    std::cout << "blocks: " << blocks << " threads: " << NUM_THREADS << std::endl;

    thrust::host_vector<int> h_vec(NUM_COUNT);
    thrust::fill(h_vec.begin(), h_vec.end(), 1);

    thrust::device_vector<int> d_vec(NUM_COUNT);
    thrust::device_vector<int> d_answer(1);
    thrust::device_vector<int> d_partial(blocks);

    thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

    int *raw_point_nums = thrust::raw_pointer_cast(&d_vec[0]);
    int *raw_point_partial = thrust::raw_pointer_cast(&d_partial[0]);
    int *raw_point_answer = thrust::raw_pointer_cast(&d_answer[0]);

    struct
    {
        std::string name;
        pfnReduction func;
    } rgTests[] = {
        { "simple loop", reduction1 },
        { "simple loop template", reduction1t },
        { "atomicAdd", reduction2 },
        { "atomicAdd template", reduction2t },
        { "single pass", reduction3 },
        { "single pass template", reduction3t },        
    };

    int numTests = sizeof(rgTests) / sizeof(rgTests[0]);
    int host_answer = thrust::reduce(h_vec.begin(), h_vec.end());
    for (int i = 0; i < numTests; ++i)
    {
        double time = run(100, raw_point_answer, raw_point_partial, 
                          raw_point_nums, NUM_COUNT, blocks, 
                          NUM_THREADS, rgTests[i].func);
        int h_answer = d_answer[0];
        
        std::string equal = (host_answer == h_answer) ? "=" : "!=";

        std::cout << rgTests[i].name <<  " time: " << time 
                  << "ms host answer (" << host_answer << ") " 
                  << equal << " device answer (" << h_answer << ")" 
                  << std::endl;
    }

    return 0;
}